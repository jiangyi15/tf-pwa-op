
#include <hip/hip_runtime.h>
/* Copyright 2019 The TensorFlow Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/

#if GOOGLE_CUDA

#define EIGEN_USE_GPU

#include "small_d.h"
#include "tensorflow/core/util/gpu_kernel_helper.h"
#include <assert.h>

namespace tensorflow {
namespace functor {

typedef Eigen::GpuDevice GPUDevice;

// Define the CUDA kernel.
//template <typename T>
//__global__ void SmallDSinCosCudaKernel(const int size, const int j, const T* beta, T* sincos) {
//  auto n = (j+1);
//  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size;
//       i += blockDim.x * gridDim.x) {
//        T sa = sin(beta[i]/2);
//        T ca = cos(beta[i]/2);
//        for (int l=0;l<n;l++){
//            sincos[i*n + l] = pow(sa,l) *pow(ca, j-l);
//        }
//    }
//}
//
//template <typename T>
//__global__ void SmallDCudaKernel(const int size, const int j, const T* w, const T* sincos, T* out) {
//  auto n = (j+1);
//  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size;
//       i += blockDim.x * gridDim.x) {
//        for (int j1=0;j1<n;j1++){
//        for (int j2=0;j2<n;j2++){
//            out[i*n*n + j1*n + j2] = 0.0;
//        for (int l=0;l<j+1;l++){
//            out[i*n*n + j1*n + j2] += w[l*n*n+j1*n+j2] * sincos[i*n+l];
//        }
//        }
//        }
//  }
//}

// Define the CUDA kernel.
template <typename T>
__global__ void SmallDSinCosCudaKernel(const int size, const int j, const T* beta, T* sincos) {
  auto n = (j+1);
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int l = tid % n;
  int b_id = tid / n;

  if (tid < size * n) {
      T b = beta[b_id];
      T sa = sin(b/2);
      T ca = cos(b/2);

      sincos[b_id * n + l] = pow(sa,l) * pow(ca, j-l);
  }
}

template <typename T>
__global__ void SmallDCudaKernel(const int size, const int j, const T* w, const T* sincos, T* out) {
  auto n = (j+1);
  int j2 = threadIdx.x % n;
  int j1 = threadIdx.x / n;
  int i = blockIdx.x;

  T sum  = 0.0;

  for (int l=0;l<j+1;l++){
      sum += w[l*n*n+j1*n+j2] * sincos[i*n+l];
  }

      out[i*n*n + j1*n + j2] = sum;
}


// Define the GPU implementation that launches the CUDA kernel.
template <typename T>
struct SmallDFunctor<GPUDevice, T> {
  void operator()(const GPUDevice& d, int size, int j, const T* in, const T* w,T* sincos, T* out) {
    // Launch the cuda kernel.
    //
    // See core/util/cuda_kernel_helper.h for example of computing
    // block count and thread_per_block count.
    //int block_count = 1024;
    //int thread_per_block = 20;
    //SmallDSinCosCudaKernel<T>
    //    <<<block_count, thread_per_block, 0, d.stream()>>>(size, j, in, sincos);
    //SmallDCudaKernel<T>
    //    <<<block_count, thread_per_block, 0, d.stream()>>>(size, j, w, sincos, out);
    int block_count = (size * (j + 1) + 511) / 512;
    int thread_per_block = 512;
    SmallDSinCosCudaKernel
        <<<block_count, thread_per_block>>>(size, j, in, sincos);

    assert((j+1) * (j+1) <= 1024);
    block_count = size;
    thread_per_block = (j+1) * (j+1);
    SmallDCudaKernel
        <<<block_count, thread_per_block>>>(size, j, w, sincos, out);

  }
};

// Explicitly instantiate functors for the types of OpKernels registered.
template struct SmallDFunctor<GPUDevice, float>;
template struct SmallDFunctor<GPUDevice, double>;
}  // end namespace functor
}  // end namespace tensorflow

#endif  // GOOGLE_CUDA
